#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <bits/stdc++.h> 
#include <random>
#include <algorithm>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/extrema.h>

using namespace std;
using namespace thrust;

struct custom_transform
{
   
    thrust::device_ptr<double> flat_mat;
    int *pontos;
    int n, seeds;

    custom_transform (thrust::device_ptr<double> flat_mat,int *pontos, int n, int seeds) : flat_mat(flat_mat), pontos(pontos), n(n), seeds(seeds) {}; 

    __host__ __device__
        double operator()(const int& x) {

            thrust::minstd_rand rng(x*1331+seeds);
            thrust::uniform_int_distribution<int> dist(0,n-1);
            
            double local_dist = 0;
            
            for (int i = 0; i < n; i++)
            {
                pontos[(n*x)+i] = i;
            }
            
            int v;
            int temp;  

            for (v = n-1; v > 0; --v) {
                int rnd = dist(rng);
                temp = pontos[(n*x)+v];
                pontos[(n*x)+v] = pontos[(n*x)+rnd];
                pontos[(n*x)+rnd] = temp;
            }
            
            for (int h = 1; h < n; h++){
                local_dist += flat_mat[pontos[(n*x)+h-1] * n + pontos[(n*x)+h]];
            }                
            local_dist += flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+0]];

            int i = 0;
            bool trocou = false;
            
            for (i = 0; i < n; i++)
            {    
                for (int j = i+1; j < n; j++)
                {
                    double dist = 0; 
                    double comp1 = 0;
                    double comp2 = 0;

                    if (i == 0){
                        if (j == n - 1){
                            comp1 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        } else {
                        comp1 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        }
                    } else if (j == 0){
                        if (i == n - 1){
                            comp1 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        } else {
                        comp1 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];    
                        }
                    } else if (j == n - 1){
                        if (i == 0){
                            comp1 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        } else {
                        comp1 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        }
                    } else if (i == n - 1){
                        if (j == 0){
                            comp1 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        } else {
                        comp1 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        }
                    } else {
                        comp1 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];                     
                    }
                    
                    thrust::swap(pontos[(n*x)+i], pontos[(n*x)+j]);
             
                    if (i == 0){
                        if (j == n - 1){
                            comp2 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        } else {
                        comp2 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        }
                    } else if (j == 0){
                        if (i == n - 1){
                            comp2 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        } else {
                        comp2 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];    
                        }
                    } else if (j == n - 1){
                        if (i == 0){
                            comp2 = flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        } else {
                        comp2 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+0]];
                        }
                    } else if (i == n - 1){
                        if (j == 0){
                            comp2 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        } else {
                        comp2 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+0]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];
                        }
                    } else {
                        comp2 = flat_mat[pontos[(n*x)+i-1] * n + pontos[(n*x)+i]] + flat_mat[pontos[(n*x)+i] * n + pontos[(n*x)+i+1]] + flat_mat[pontos[(n*x)+j-1] * n + pontos[(n*x)+j]] + flat_mat[pontos[(n*x)+j] * n + pontos[(n*x)+j+1]];                     
                    }
                        
                    if (comp2 < comp1) {
                        for (int h = 1; h < n; h++){
                        dist += flat_mat[pontos[(n*x)+h-1] * n + pontos[(n*x)+h]];                            
                        }                
                        dist += flat_mat[pontos[(n*x)+n-1] * n + pontos[(n*x)+0]];          
                        trocou = true;
                        local_dist = dist;
                        break;    
                    } else { 
                        thrust::swap(pontos[(n*x)+j], pontos[(n*x)+i]);
                    }
                }
                if (trocou && i == n - 1){
                    trocou = false;
                    i = -1;
                }
            }
            return local_dist;
        }
};

int main(){
    int seeds;
    
    if (getenv("SEED") == NULL)
        seeds = 10;
    else
        seeds = atoi(getenv("SEED"));

    vector<double> x, y, flat_mat1;
    double xi, yi;
    int n;

    cin >> n;

    for (int i = 0; i < n; i++)
    {
        cin >> xi >> yi;
        x.push_back(xi);
        y.push_back(yi);
    }
    
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            flat_mat1.push_back(sqrt(pow(x[i] - x[j], 2) + pow(y[i] - y[j], 2)));
        }
    }

    thrust::device_vector<double> flat_mat(flat_mat1);
    
    thrust::counting_iterator<int> iter(0);
    
    device_vector<double> vec(10*n, 0.0);    
    
    thrust::device_vector<int> pontos(10*n*n, 0);

    custom_transform cus(thrust::device_ptr<double>(flat_mat.data()), thrust::raw_pointer_cast(pontos.data()), n, seeds);
    
    thrust::transform(iter, iter + 10*n, vec.begin(), cus);
    
    thrust::device_vector<double>::iterator iter2 = thrust::min_element(vec.begin(), vec.end());

    int position = iter2 - vec.begin();
    
    if (getenv("DEBUG")){
        for (int i = 0; i < vec.size(); i++){
            cerr << "local: " << vec[i] << " ";
            for (int j = 0; j < n; j++)
            {
                cerr << pontos[i*n+j] << " ";
            }
            cerr << endl;
        }
    }
    
    cout << vec[position] << " " << 0 << endl;
    
    for (int i = 0; i < n; i++)
    {
        cout << pontos[position*n+i] << " ";
    }
    
    cout << endl;

    return 0;
}
